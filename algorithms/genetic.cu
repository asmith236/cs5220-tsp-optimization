#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <vector>
#include <iostream>
#include <numeric>
#include <random>
#include <cassert>
#include "../common/algorithms.hpp" // Include the common header for TSPResult

using namespace std;

#define BLOCK_SIZE 256
#define DEBUG 0 // Debug flag for verbose output

// Device function to calculate the Euclidean distance between two points
__device__ double distance(const double2& p1, const double2& p2) {
    return sqrt((p1.x - p2.x) * (p1.x - p2.x) + (p1.y - p2.y) * (p1.y - p2.y));
}

// Kernel to initialize the distance matrix
__global__ void initDistanceMatrix(const double2* coordinates, double* distances, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n) {
        int i = idx / n;
        int j = idx % n;
        distances[idx] = distance(coordinates[i], coordinates[j]);
    }
}

// Kernel to calculate fitness values for a population
__global__ void calculateFitness(const int* population, const double* distances, double* fitness, int n, int pop_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < pop_size) {
        const int* route = &population[idx * n];
        double route_distance = 0.0;

        // Calculate the total distance of the route
        for (int i = 0; i < n - 1; ++i) {
            int from = route[i];
            int to = route[i + 1];
            route_distance += distances[from * n + to];
        }
        route_distance += distances[route[n - 1] * n + route[0]]; // Closing the loop

        // Avoid divide by zero
        if (route_distance <= 0.0) {
            fitness[idx] = 0.0;
        } else {
            fitness[idx] = 1.0 / route_distance;
        }

        // Debug: Print route distance
        if (DEBUG && threadIdx.x == 0) {
            printf("Route %d: Distance = %.6f, Fitness = %.6f\n", idx, route_distance, fitness[idx]);
        }
    }
}

// Device function to repair invalid routes
__device__ void rebuildRoute(int* route, int n) {
    bool visited[256] = {false};

    // Mark all valid and visited nodes
    for (int i = 0; i < n; ++i) {
        if (route[i] >= 0 && route[i] < n) {
            if (!visited[route[i]]) {
                visited[route[i]] = true;
            } else {
                route[i] = -1; // Mark duplicates as invalid
            }
        } else {
            route[i] = -1; // Mark invalid entries
        }
    }

    // Replace invalid or duplicate nodes
    int missing_idx = 0;
    for (int i = 0; i < n; ++i) {
        if (route[i] == -1) {
            while (missing_idx < n && visited[missing_idx]) {
                ++missing_idx;
            }
            route[i] = missing_idx;
            visited[missing_idx] = true;
        }
    }
}

// Kernel for mutation with optional debugging and repair
__global__ void mutatePopulation(int* population, int pop_size, int n, double mutation_rate, int seed, bool debug) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed + idx, 0, 0, &state);

    if (idx < pop_size) {
        int* route = &population[idx * n];

        if (debug) {
            printf("Initial route for index %d: ", idx);
            for (int i = 0; i < n; ++i) {
                printf("%d ", route[i]);
            }
            printf("\n");
        }

        if (hiprand_uniform(&state) < mutation_rate) {
            int i = hiprand(&state) % n;
            int j = hiprand(&state) % n;
            while (j == i) {
                j = hiprand(&state) % n;
            }
            int temp = route[i];
            route[i] = route[j];
            route[j] = temp;
        }

        rebuildRoute(route, n);

        if (debug) {
            printf("Mutated and repaired route for index %d: ", idx);
            for (int i = 0; i < n; ++i) {
                printf("%d ", route[i]);
            }
            printf("\n");
        }
    }
}

// Function to generate random initial population on the host
vector<vector<int>> generateInitialPopulation(int pop_size, int n) {
    vector<vector<int>> population(pop_size, vector<int>(n));
    random_device rd;
    mt19937 g(rd());

    for (int i = 0; i < pop_size; ++i) {
        iota(population[i].begin(), population[i].end(), 0);
        shuffle(population[i].begin(), population[i].end(), g);
    }
    return population;
}

// Function to debug the initial population on the device
void debugInitialPopulationOnDevice(int* d_population, int pop_size, int n) {
    vector<int> host_population(pop_size * n);
    hipMemcpy(host_population.data(), d_population, pop_size * n * sizeof(int), hipMemcpyDeviceToHost);

    cout << "Initial population on device:" << endl;
    for (int i = 0; i < pop_size; ++i) {
        cout << "Route " << i << ": ";
        for (int j = 0; j < n; ++j) {
            cout << host_population[i * n + j] << " ";
        }
        cout << endl;
    }
}

// Main solve function using CUDA
TSPResult solve(const vector<pair<double, double>>& coordinates) {
    int n = coordinates.size();
    int pop_size = 100, generations = 300;
    double mutation_rate = 0.01;

    // Convert coordinates to double2
    vector<double2> host_coordinates(n);
    for (int i = 0; i < n; ++i) {
        host_coordinates[i] = {coordinates[i].first, coordinates[i].second};
    }

    // Allocate device memory
    double2* d_coordinates;
    double* d_distances;
    int* d_population;
    double* d_fitness;

    hipMalloc(&d_coordinates, n * sizeof(double2));
    hipMalloc(&d_distances, n * n * sizeof(double));
    hipMalloc(&d_population, pop_size * n * sizeof(int));
    hipMalloc(&d_fitness, pop_size * sizeof(double));

    hipMemcpy(d_coordinates, host_coordinates.data(), n * sizeof(double2), hipMemcpyHostToDevice);

    // Initialize distance matrix
    initDistanceMatrix<<<(n * n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_coordinates, d_distances, n);
    hipDeviceSynchronize();

    // Generate and copy initial population
    auto host_population = generateInitialPopulation(pop_size, n);
    vector<int> flat_population;
    for (const auto& individual : host_population) {
        flat_population.insert(flat_population.end(), individual.begin(), individual.end());
    }
    hipMemcpy(d_population, flat_population.data(), pop_size * n * sizeof(int), hipMemcpyDeviceToHost);

    if (DEBUG) {
        debugInitialPopulationOnDevice(d_population, pop_size, n);
    }

    // Evolution loop
    vector<int> best_route;
    double best_distance = numeric_limits<double>::infinity();

    for (int gen = 0; gen < generations; ++gen) {
        calculateFitness<<<(pop_size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
            d_population, d_distances, d_fitness, n, pop_size);
        hipDeviceSynchronize();

        thrust::device_ptr<double> fitness_ptr(d_fitness);
        thrust::device_ptr<int> population_ptr(d_population);

        // Ensure sorting is correct (lower distance -> higher fitness)
        thrust::sort_by_key(fitness_ptr, fitness_ptr + pop_size, population_ptr, thrust::greater<double>());

        mutatePopulation<<<(pop_size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
            d_population, pop_size, n, mutation_rate, gen, DEBUG);
        hipDeviceSynchronize();

        if (DEBUG) {
            debugInitialPopulationOnDevice(d_population, pop_size, n);
        }

        vector<int> current_best_route(n);
        hipMemcpy(current_best_route.data(), d_population, n * sizeof(int), hipMemcpyDeviceToHost);

        double current_best_distance;
        hipMemcpy(&current_best_distance, d_fitness, sizeof(double), hipMemcpyDeviceToHost);

        double route_distance = 1.0 / current_best_distance; // Convert fitness back to distance
        if (route_distance < best_distance) {
            best_distance = route_distance;
            best_route = current_best_route;
        }

        if (DEBUG) {
            cout << "Generation " << gen << ": Best Distance = " << best_distance << endl;
        }
    }

    hipFree(d_coordinates);
    hipFree(d_distances);
    hipFree(d_population);
    hipFree(d_fitness);

    TSPResult result;
    result.cost = best_distance;
    result.path = best_route;
    return result;
}
