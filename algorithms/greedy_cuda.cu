#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "../common/algorithms.hpp"

const int BLOCK_SIZE = 256;

__global__ void findNearestCityKernel(
    const double* coords_x, // Array of x coordinates
    const double* coords_y, // Array of y coordinates
    const char* visited, // Array of visited cities (1 if visited, 0 if not)
    int current_city, // idx of the current city
    int n, // Total number of cities
    double* min_distances, //Output: minimum distance to each city
    int* next_cities // Output: next city to visit
) {
    __shared__ double shared_min_distances[BLOCK_SIZE]; // Shared memory for minimum distances
    __shared__ int shared_next_cities[BLOCK_SIZE]; // Shared memory for next cities
    
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load current city coordinates into shared memory
    __shared__ double current_coords[2];
    if (tid == 0) { // Only the first thread in the block loads the current city coordinates
        current_coords[0] = coords_x[current_city];
        current_coords[1] = coords_y[current_city];
    }
    __syncthreads(); // Ensures all threads wait until the coordinates are loaded
    
    double local_min = INFINITY;
    int local_next = -1;
    
    // Each thread maintains its own minimum distance and corresponding city
    for (int j = gid; j < n; j += blockDim.x * gridDim.x) { //blockDim.x * gridDim.x is the total number of threads
        if (!visited[j]) { // If the city has not been visited
            double dx = current_coords[0] - coords_x[j];
            double dy = current_coords[1] - coords_y[j];
            double dist = sqrt(dx * dx + dy * dy);
            if (dist < local_min) { // If the distance is less than the local minimum
                local_min = dist;
                local_next = j;
            }
        }
    }

    // For example, if n = 1000, and we have 256 threads, then each thread will have to calculate 4 cities
    
    shared_min_distances[tid] = local_min;
    shared_next_cities[tid] = local_next;
    __syncthreads(); // Ensures all threads wait until the minimum distances and next cities are loaded
    
    // Block that performs parallel reduction to find the global minimum distance and corresponding city

    // Explanation of what is parallel reduction:
    /* In parallel reduction, we are trying to find the minimum distance and the corresponding city from the shared memory
    array shared_min_distances. for example, if we have 256 threads, then we have 256 elements in shared_min_distances.
    We are trying to find the minimum distance and the corresponding city from these 256 elements. 
    the stride initially is 128, then 64, then 32, then 16, then 8, then 4, then 2, then 1.
    so we are comparing the elements at index 0 and 128 (along with 1 and 129, 2 and 130 etc etc), then 0 and 64, then 0 and 32, then 0 and 16, then 0 and 8, then 0 and 4, then 0 and 2, then 0 and 1.
    and we are updating the minimum distance and the corresponding city accordingly.
    */
    for (int stride = BLOCK_SIZE/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            if (shared_min_distances[tid + stride] < shared_min_distances[tid]) {
                shared_min_distances[tid] = shared_min_distances[tid + stride];
                shared_next_cities[tid] = shared_next_cities[tid + stride];
            }
        }
        __syncthreads();
    }
    
    // After the parallel reduction, the minimum distance and the corresponding city will be at index 0
    if (tid == 0) {
        min_distances[blockIdx.x] = shared_min_distances[0];
        next_cities[blockIdx.x] = shared_next_cities[0];
    }

    /* Benefits of parallel reduction:
    1. Reduces complexity from O(n) to O(log(n))
    2. Reduces the number of global memory accesses
    */
}

TSPResult solve(const std::vector<std::pair<double, double>>& coordinates) {
    int n = coordinates.size();
    
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem); // Command that gets the available free memory on the GPU
    
    // Pinned memory: memory that is not swapped out to disk optimizes memory transfer between CPU and GPU
    // Putting x and y coordinates into pinned memory
    double *h_coords_x, *h_coords_y;
    hipHostMalloc(&h_coords_x, n * sizeof(double));
    hipHostMalloc(&h_coords_y, n * sizeof(double));
    
    for (int i = 0; i < n; i++) {
        h_coords_x[i] = coordinates[i].first;
        h_coords_y[i] = coordinates[i].second;
    }
    
    /* Allocating GPU memory for coordaiantes.
    Copying data from CPU to GPU*/
    double *d_coords_x, *d_coords_y;
    hipMalloc(&d_coords_x, n * sizeof(double));
    hipMalloc(&d_coords_y, n * sizeof(double));
    
    hipMemcpy(d_coords_x, h_coords_x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_coords_y, h_coords_y, n * sizeof(double), hipMemcpyHostToDevice);
    
    // Initialize path construction variables
    std::vector<int> path;
    double total_cost = 0.0;
    std::vector<char> visited(n, 0);  // Using char instead of bool
    
    // Allocate device memory for visited array
    char* d_visited;
    hipMalloc(&d_visited, n * sizeof(char));
    
    // Start from city 0
    int current_city = 0;
    path.push_back(current_city);
    visited[current_city] = 1;  // 1 stands for true, means that city has been visited.
    hipMemcpy(d_visited, visited.data(), n * sizeof(char), hipMemcpyHostToDevice);
    
    // Allocate device memory for minimum distance search
    double* d_min_distances;
    int* d_next_cities;
    hipMalloc(&d_min_distances, BLOCK_SIZE * sizeof(double)); // We are going to use BLOCK_SIZE threads
    hipMalloc(&d_next_cities, BLOCK_SIZE * sizeof(int)); 
    
    const int threads_per_block = BLOCK_SIZE;
    const int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    // Main loop
    while (path.size() < n) {
        // Launch kernel to find nearest city
        findNearestCityKernel<<<BLOCK_SIZE, BLOCK_SIZE>>>(
            d_coords_x,
            d_coords_y,
            d_visited,
            current_city,
            n,
            d_min_distances,
            d_next_cities
        );
        
        // Copy results back to host
        std::vector<double> h_min_distances(BLOCK_SIZE);
        std::vector<int> h_next_cities(BLOCK_SIZE);
        hipMemcpy(h_min_distances.data(), d_min_distances, BLOCK_SIZE * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_next_cities.data(), d_next_cities, BLOCK_SIZE * sizeof(int), hipMemcpyDeviceToHost);
        
        // Find global minimum
        double min_distance = INFINITY;
        int next_city = -1;
        for (int i = 0; i < BLOCK_SIZE; i++) {
            if (h_min_distances[i] < min_distance) {
                min_distance = h_min_distances[i];
                next_city = h_next_cities[i];
            }
        }
        
        // Update path
        current_city = next_city;
        path.push_back(current_city);
        visited[current_city] = 1;  // Using 1 instead of true
        hipMemcpy(d_visited, visited.data(), n * sizeof(char), hipMemcpyHostToDevice);
        total_cost += min_distance;
    }
    
    // Calculate distance back to start
    double final_distance;
    double dx = h_coords_x[path.back()] - h_coords_x[path[0]];
    double dy = h_coords_y[path.back()] - h_coords_y[path[0]];
    final_distance = sqrt(dx * dx + dy * dy);
    total_cost += final_distance;
    
    // Cleaning up all the variables that we have used
    hipHostFree(h_coords_x);
    hipHostFree(h_coords_y);
    hipFree(d_coords_x);
    hipFree(d_coords_y);
    hipFree(d_visited);
    hipFree(d_min_distances);
    hipFree(d_next_cities);
    
    TSPResult result;
    result.cost = total_cost;
    result.path = path;
    return result;
}
